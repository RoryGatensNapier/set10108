#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <>

#define STB_IMAGE_IMPLEMENTATION
#include <stb_image.h>

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include <stb_image_write.h>

using namespace std;

// 5x5 gaussian kernel radius
constexpr int RADIUS = 2;

__global__ void smooth(float * values_in, float * values_out, int width, int height)
{
	// 5x5 Gaussian kernel. It's 1D and has 25 elements, although here it's made to look like 2D because of the newlines
	constexpr float KERNEL[] = {
		0.003765, 	0.015019, 	0.023792, 	0.015019, 	0.003765,
		0.015019, 	0.059912, 	0.094907, 	0.059912, 	0.015019,
		0.023792, 	0.094907, 	0.150342, 	0.094907, 	0.023792,
		0.015019, 	0.059912, 	0.094907, 	0.059912, 	0.015019,
		0.003765, 	0.015019, 	0.023792, 	0.015019, 	0.003765,
	};

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	// prevent out-of-bounds
	if (x >= width || y >= height)
		return;

	// temporary variable to put the weighted sum
	float tmp = 0.0f;
	for (int oy = -RADIUS; oy <= RADIUS; ++oy)
	{
		// clamp the value to the boundary value. Instead of going out of bounds, we're going to be using the last element possible.
		int ynb = min(max(y + oy,0), height-1);
		int kernel_row = oy + RADIUS; // in the range of [0,2*RADIUS]
		for (int ox = -RADIUS; ox <= RADIUS; ++ox)
		{
			int xnb = min(max(x + ox,0),width-1);
			int kernel_col = ox + RADIUS;// in the range of [0,2*RADIUS]
			float value = values_in[xnb + ynb * width] * KERNEL[kernel_col + kernel_row*5];
			tmp += value;
		}
	}
	values_out[x + y * width] = tmp;
}

std::vector<float> load_image_to_grayscale(const char* filename, int& width, int& height)
{
	int n;
	auto imgdata = (uint8_t*)stbi_load(filename, &width, &height, &n, 0); 
	vector<float> values(width*height);
	for(int y=0;y<height; ++y)
		for (int x = 0; x < width; ++x)
		{
			int o = x + y * width; // 1d index
			// is it RGB or RGBA?
			if (n >= 3)
			{
				float r = imgdata[o * n] / 255.0f;
				float g = imgdata[o * n+1] / 255.0f;
				float b = imgdata[o * n+2] / 255.0f;
				values[o] = 0.2126f * r + 0.7152f * g + 0.0722f * b; // RGB -> grayscale conversion formula
			}
			else if (n == 1)
			{
				values[o] = imgdata[o]/255.0f;
			}
		}
	stbi_image_free(imgdata);
	return values;
}

void save_grayscale_png(const char* filename, const std::vector<float>& values, int width, int height)
{
	std::vector<uint8_t> imgdata(width * height);
	for (int y = 0; y < height; ++y)
		for (int x = 0; x < width; ++x)
			imgdata[x + y * width] = uint8_t(values[x + y * width] * 255);
	stbi_write_png(filename, width, height, 1, imgdata.data(), width);
}

int main(int argc, char **argv)
{
	//if (argc == 1)
	//{
	//	printf("Please provide an image filename as the argument (full path). The program will create a new image with the suffix _out.png and will place it alongside the original image");
	//	exit(0);
	//}
	int width, int height;
	std::string filename_in = "D:/CPS_CW1_IMG/image_fever_example/labs/test.jpg";
	auto h_values_in = load_image_to_grayscale(filename_in.c_str(), width, height);
	const size_t NUM_PIXELS = h_values_in.size();

	// Create host memory
	const size_t NUM_BYTES = sizeof(float) * NUM_PIXELS;
	vector<float> h_values_out(NUM_PIXELS);    // Output array

	float* d_values_in = nullptr;
	float* d_values_out = nullptr;

	// Initialise buffers
	hipMalloc((void**)&d_values_in, NUM_BYTES);
	hipMalloc((void**)&d_values_out, NUM_BYTES);

	// Write host data to device
	hipMemcpy(d_values_in, h_values_in.data(), NUM_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(d_values_out, h_values_out.data(), NUM_BYTES, hipMemcpyHostToDevice);

	// arbitrary! Change it to something else. Is it better? Worse?
	dim3 blockDim = { 32,32,1 };
	// This is written to accomodate the blockDim we specified and the width/height of the image.
	// If unsure why the +31, check the lecture. We need to make sure we have enough threads, but not blockDim or more
	dim3 gridDim = { 
		(unsigned(width) + 31) / 32, 
		(unsigned(height) + 31) / 32, 
		1 
	};
	smooth <<<gridDim, blockDim >>>(d_values_in, d_values_out, width, height);

	// Read output buffer back to the host
	hipMemcpy(h_values_out.data(), d_values_out, NUM_BYTES, hipMemcpyDeviceToHost);

	// Clean up resources
	hipFree(d_values_in);
	hipFree(d_values_out);

	auto filename_out = filename_in.substr(0, filename_in.size() - 4) + "_out.png";
	save_grayscale_png(filename_out.c_str(), h_values_out, width, height);

	return 0;
}
